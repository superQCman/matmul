#include "hip/hip_runtime.h"
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <sys/time.h>

#include <fstream>
#include <iostream>
#include <string>

#include "apis_cu.h"
#include "hip/hip_runtime.h"
#include ""

/**
 * 本示例程序为：通过4个GPU chiplet
 * 计算随机数矩阵A（400 * 100）与随机数矩阵B（100 * 400）相乘结果。
 * 由矩阵乘法原理可知，我们可将计算任务划分为4个100*100的矩阵相乘，并将结果相加。
 */

#define Row 100
#define Col 100

/**
 * 矩阵乘法的核心函数，由每个线程都会运行一次本函数，
 * 根据线程编号不同计算出位于结果矩阵不同位置的数据。
 */

__global__ void matrix_mul_gpu(int64_t* M, int64_t* N, int64_t* P, int width) {
    int sumNum = threadIdx.x + threadIdx.y * 10;
    int i = threadIdx.x;
    int j = threadIdx.y;
    int64_t sum = 0;
    for (int k = 0; k < width; k++) {
        int64_t a = M[j * width + k];
        int64_t b = N[k * width + i];
        sum += a * b;
    }
    P[sumNum] = sum;
}

int main(int argc, char** argv) {
    // 读取本进程所代表的chiplet编号

    int idX = atoi(argv[1]);
    int idY = atoi(argv[2]);
    int64_t *d_dataA, *d_dataB, *d_dataC;
    hipMalloc((void**)&d_dataA, sizeof(int64_t) * Row * Col);
    hipMalloc((void**)&d_dataB, sizeof(int64_t) * Row * Col);
    hipMalloc((void**)&d_dataC, sizeof(int64_t) * Col * Row);

    receiveMessage(idX, idY, 3, 3, d_dataA, sizeof(int64_t) * Row * Col);
    receiveMessage(idX, idY, 3, 3, d_dataB, sizeof(int64_t) * Row * Col);

    // calculate
    dim3 threadPerBlock(10, 10);
    dim3 blockNumber(1);
    matrix_mul_gpu<<<blockNumber, threadPerBlock>>>(d_dataA, d_dataB, d_dataC, Col);

    sendMessage(3, 3, idX, idY, d_dataC, Row * Col * sizeof(int64_t));
    hipFree(d_dataA);
    hipFree(d_dataB);
    hipFree(d_dataC);
    return 0;
}
